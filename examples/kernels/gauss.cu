#include "hip/hip_runtime.h"
#include "examples/kernels/gauss.h"

extern "C" __global__ void gaussFilterKernel(Pixel *image,
                                             float weight[5][5],
                                             int width, int height) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  Pixel new_value;
  new_value.r = 0;
  new_value.g = 0;
  new_value.b = 0;
  for (int xl = -2; xl <= 2; ++xl) {
    for (int yl = -2; yl <= 2; ++yl) {
      if (((col + xl) + (row + yl) * width) < 0 ||
          ((col + xl) + (row + yl) * width) >= width * height) {
        continue;
      }
      new_value.r +=
          image[(col + xl) + (row + yl) * width].r * weight[xl + 2][yl + 2];
      new_value.g +=
          image[(col + xl) + (row + yl) * width].g * weight[xl + 2][yl + 2];
      new_value.b +=
          image[(col + xl) + (row + yl) * width].b * weight[xl + 2][yl + 2];
    }
  }
  image[col + row * width] = new_value;
}